#include "hip/hip_runtime.h"
#include "CudaFunctions.cuh"
#include "Simulation.h"
#include "Constants.h"

void MyCudaWrapper::Init(Simulation& simulation) {
	// Alloc particles on GPU
	CHECK_ERROR(hipMalloc((void**)&simulation.particlesDevice, (simulation.MAX_PARTICLES) * sizeof(Particle)));

	// Page lock host particles
	size_t memsize = ((simulation.MAX_PARTICLES * sizeof(Particle) + 4095) / 4096) * 4096;
	//CHECK_ERROR(hipHostRegister(simulation.particles.data(), memsize, hipHostRegisterMapped));

	// Move particles from host to device
	CopyParticlesHostToDevice(simulation);

	// Allocate space for spatial grid
	CHECK_ERROR(hipMalloc((void**)&simulation.particleGrid.gridDevice, simulation.particleGrid.grid.size() * sizeof(int)));

	std::cout << "Allocated " << simulation.particleGrid.grid.size() * sizeof(int) << "for grid" << std::endl;

	// Page-lock host location of spatial grid
	// TODO: fix :(
	//memsize = ((simulation.particleGrid.grid.size() * sizeof(int) + 4095) / 4096) * 4096;
	//CHECK_ERROR(hipHostRegister(simulation.particleGrid.grid.data(), memsize, hipHostRegisterMapped));

	// Move spatial grid to GPU
	CopyGridHostToDevice(simulation);

	CHECK_ERROR(hipDeviceSynchronize());
	std::cout << "GPU initialized" << std::endl;
}

void MyCudaWrapper::Update(Simulation& simulation, float timeStep) {
	CHECK_ERROR(hipDeviceSynchronize());
	// copy spatial grid to device
	CopyGridHostToDevice(simulation);

	// call kernels
	unsigned int blockCount = std::ceil(simulation.particleCount / 256);
	densityPressureKernel << <blockCount, 256 >> > (simulation.particleCount, simulation.particlesDevice, simulation.particleGrid, simulation.MASS, simulation.GAS_CONST, simulation.REST_DENS);
	forceKernel << <blockCount, 256 >> > (simulation.particleCount, simulation.particlesDevice, simulation.particleGrid, simulation.MASS, simulation.VISC, simulation.G);
	integrateKernel << <blockCount, 256 >> > (simulation.particleCount, simulation.particlesDevice, timeStep, simulation.BOUND_DAMPING, simulation.VIEW_WIDTH, simulation.VIEW_HEIGHT);


	// copy particles back to host
	CopyParticlesDeviceToHost(simulation);
}

void MyCudaWrapper::CopyParticlesHostToDevice(Simulation& simulation) {
	CHECK_ERROR(hipMemcpy(simulation.particlesDevice, simulation.particles.data(), simulation.particleCount * sizeof(Particle), hipMemcpyHostToDevice));
}

void MyCudaWrapper::CopyParticlesDeviceToHost(Simulation& simulation) {
	CHECK_ERROR(hipMemcpy(simulation.particles.data(), simulation.particlesDevice, simulation.particleCount * sizeof(Particle), hipMemcpyDeviceToHost));
}

void MyCudaWrapper::CopyGridHostToDevice(Simulation& simulation) {
	CHECK_ERROR(hipMemcpy(simulation.particleGrid.gridDevice, simulation.particleGrid.grid.data(), simulation.particleGrid.grid.size() * sizeof(int), hipMemcpyHostToDevice));
}

__global__ void densityPressureKernel(int particleCount, Particle* particles, ParticleGrid grid, float MASS, float GAS_CONST, float REST_DENS)
{
	int particleID = blockDim.x * blockIdx.x + threadIdx.x;
	if (particleID > particleCount) {
		return;
	}

	Particle& pi = particles[particleID];
	pi.rho = 0.0f;

	int posX;
	int posY;
	grid.Index1Dto2D(pi.id, posX, posY);

	// Go over neighbour cells
	for (int x = posX - 1; x < posX + 1; x++) {
		for (int y = posY - 1; y < posY + 1; y++) {
			// Check grid boundaries
			if (x < 0 || x >= grid.dimX || y < 0 || y >= grid.dimY) {
				continue;
			}

			int gridIndex = grid.Index2Dto1D(x, y);
			int currentIndex = grid.gridDevice[gridIndex];

			// While there are some neighbours in that grid cell
			while (currentIndex != -1) {
				Particle& pj = particles[currentIndex];
				
				// TODO: check wasnt there
				if (pi.id != pj.id) {
					MyVec2 rij = pj.position - pi.position;
					float r2 = rij.LengthSquared();

					if (r2 < HSQ) {
						pi.rho += MASS * POLY6 * pow(HSQ - r2, 3.f);
					}
				}

				currentIndex = pj.nextParticle;
			}
		}
	}
	pi.p = GAS_CONST * (pi.rho - REST_DENS);
}

__global__ void forceKernel(int particleCount, Particle* particles, ParticleGrid grid, float MASS, float VISC, MyVec2 G)
{
	/*
		for (int i = 0; i < particleCount; i++) {
		Particle& pi = particles[i];

		MyVec2 fpress(0.f, 0.f);
		MyVec2 fvisc(0.f, 0.f);

		std::vector<int> potentialNeighbours;
		GetNeighbourParticlesIndices(pi.id, potentialNeighbours);

		for (int index : potentialNeighbours) {
			Particle pj = particles[index];
			if (pi.id == pj.id) {
				continue;
			}

			MyVec2 rij = pj.position - pi.position;
			float r = rij.Length();

			if (r < H) {
				//std::cout << "Collision in forces for id: " << pi.id << " and " << pj.id << std::endl;
				// compute pressure force contribution
				fpress += -rij.Normalized() * MASS * (pi.p + pj.p) / (2.f * pj.rho) * SPIKY_GRAD * pow(H - r, 3.f);
				// compute viscosity force contribution
				fvisc += VISC * MASS * (pj.velocity - pi.velocity) / pj.rho * VISC_LAP * (H - r);
			}
		}
		MyVec2 fgrav = G * MASS / pi.rho;
		pi.force = fpress + fvisc + fgrav;
	}
	*/
	int particleID = blockDim.x * blockIdx.x + threadIdx.x;
	if (particleID > particleCount) {
		return;
	}

	Particle& pi = particles[particleID];
	
	MyVec2 fpress(0.f, 0.f);
	MyVec2 fvisc(0.f, 0.f);

	int posX;
	int posY;
	grid.Index1Dto2D(pi.id, posX, posY);

	// Go over neighbour cells
	for (int x = posX - 1; x < posX + 1; x++) {
		for (int y = posY - 1; y < posY + 1; y++) {
			// Check grid boundaries
			if (x < 0 || x >= grid.dimX || y < 0 || y >= grid.dimY) {
				continue;
			}

			int gridIndex = grid.Index2Dto1D(x, y);
			int currentIndex = grid.gridDevice[gridIndex];

			// While there are some neighbours in that grid cell
			while (currentIndex != -1) {
				Particle& pj = particles[currentIndex];

				if (pi.id == pj.id) {
					continue;
				}

				MyVec2 rij = pj.position - pi.position;
				float r = rij.Length();

				if (r < H) {
					// compute pressure force contribution
					fpress += -rij.Normalized() * MASS * (pi.p + pj.p) / (2.f * pj.rho) * SPIKY_GRAD * pow(H - r, 3.f);
					// compute viscosity force contribution
					fvisc += VISC * MASS * (pj.velocity - pi.velocity) / pj.rho * VISC_LAP * (H - r);
				}
			}
		}
	}

	MyVec2 fgrav = G * MASS / pi.rho;
	pi.force = fpress + fvisc + fgrav;
}

__global__ void integrateKernel(int particleCount, Particle* particles, float timeStep, float BOUND_DAMPING, float VIEW_WIDTH, float VIEW_HEIGHT)
{
	int particleID = blockDim.x * blockIdx.x + threadIdx.x;
	if (particleID > particleCount) {
		return;
	}

	Particle& p = particles[particleID];

	// forward Euler integration
	if (p.rho > 0.0f) {
		p.velocity += timeStep * p.force / p.rho;
	}
	p.position += timeStep * p.velocity;

	// enforce boundary conditions
	if (p.position.x - EPS < 0.f) {
		p.velocity.x *= BOUND_DAMPING;
		p.position.x = EPS;
	}

	if (p.position.x + EPS > VIEW_WIDTH) {
		p.velocity.x *= BOUND_DAMPING;
		p.position.x = VIEW_WIDTH - EPS;
	}

	if (p.position.y - EPS < 0.f) {
		p.velocity.y *= BOUND_DAMPING;
		p.position.y = EPS;
	};

	if (p.position.y + EPS > VIEW_HEIGHT) {
		p.velocity.y *= BOUND_DAMPING;
		p.position.y = VIEW_HEIGHT - EPS;
	}
}